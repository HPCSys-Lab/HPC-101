
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

#define QTD_ELEMENTOS 1024
#define NUM_THREAD_BLOCK_Y 32
#define NUM_THREAD_BLOCK_X 32

void imprimeSoma(int *data, unsigned n)
{
    double soma = 0;
    for (int i=0; i < n; i++) {
				for (int j=0; j < n; j++){
					soma += data[i * n + j];		
				}	    
	  } 
    printf("A soma é %d\n",soma);
}

void inicializaMatriz(int *data, unsigned size)
{
  time_t t;
	srand((unsigned int) time(&t));
	for (int i=0; i<size; i++) {
     for (int j=0; j<size; j++) { 
		   //data[i * size + j] = (int)( rand() & 0xFF )/10.0f;
			 data[i * size + j] = ((int)rand() ) % 2;			 
     }  
	}
}

__global__ 
void warshallKernelShared_Principal(int *F, int k, unsigned n){
    
  //Variáveis declaradas nos registradores - visíveis somente para o thread  
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
 
  //Variáveis declaradas nos registradores - visíveis somente para o thread  
  int tidX;
  int tidY;

  //Variável compartilhada - visível somente para os threads do bloco
  __shared__ int ladrilhoPrincipal[NUM_THREAD_BLOCK_Y][NUM_THREAD_BLOCK_X];  

  //Obtém os índices dos threads
  tidX = k * blockDim.x + threadIdx.x;
  tidY = k * blockDim.y + threadIdx.y;
   
  //OBTÉM o ladrilho da DIAGONAL K (Principal)
  ladrilhoPrincipal[threadIdx.y][threadIdx.x] = F[tidY * n + tidX];
  
  //BARREIRA de sincronização - Garante que a memória compartilhada seja preenchida antes do processamento	
  __syncthreads();

  //PROCESSA o ladrilho da DIAGONAL K
  for(int m=0; m < blockDim.x; m++) {

    if(ladrilhoPrincipal[m][threadIdx.x] == 1 && ladrilhoPrincipal[threadIdx.y][m] == 1){
      ladrilhoPrincipal[threadIdx.y][threadIdx.x] = 1;   
    } 
	//BARREIRA de sincronização - Garante que as dependências de dados sejam satisfeitas. 
    __syncthreads();
  }
  //ATUALIZA os valores do ladrilho da DIAGONAL K na memória global
  F[i * n + j] = ladrilhoPrincipal[threadIdx.y][threadIdx.x];   
}

__global__ 
void warshallKernelShared_LinhaColuna(int *F, int k, unsigned n){
    
  //Variáveis declaradas nos registradores - visíveis somente para o thread	
  int tidX;
  int tidY;

  //Variáveis compartilhadas - visíveis somente para os threads do bloco
  __shared__ int ladrilhoPrincipal[NUM_THREAD_BLOCK_Y][NUM_THREAD_BLOCK_X];
  __shared__ int ladrilho[NUM_THREAD_BLOCK_Y][NUM_THREAD_BLOCK_X];
  
  //OBTÉM os índices dos threads
  tidX = k * blockDim.x + threadIdx.x;
  tidY = k * blockDim.y + threadIdx.y;

  //OBTÉM o ladrilho da DIAGONAL na memória global e atribui para a memória compartilhada
  ladrilhoPrincipal[threadIdx.y][threadIdx.x] = F[tidY * n + tidX];

  //OBTÉM os índices dos threads
  tidX = blockIdx.x * blockDim.x + threadIdx.x;
  tidY = k * blockDim.y + threadIdx.y;
  //OBTÉM o ladrilho da LINHA k na memória global e atribui para a memória compartilhada
  ladrilho[threadIdx.y][threadIdx.x] = F[tidY * n + tidX];

  //BARREIRA de sincronização - Garante que a memória compartilhada seja preenchida antes do processamento	
  __syncthreads();

  //PROCESSA o ladrilho da LINHA k
  for(int m=0; m < blockDim.x; m++) {

    if(ladrilhoPrincipal[threadIdx.y][m] == 1 && ladrilho[m][threadIdx.x] == 1){
      ladrilho[threadIdx.y][threadIdx.x] = 1;   
    }
	//BARREIRA de sincronização - Garante que as dependências de dados sejam satisfeitas. 
    __syncthreads();
  }

  //ATUALIZA os valores do ladrilho da LINHA k na memória global
  F[tidY * n + tidX] = ladrilho[threadIdx.y][threadIdx.x];   

  //===========================================================
  //OBTÉM os índices dos threads
  tidX = k * blockDim.x + threadIdx.x;
  tidY = blockIdx.x * blockDim.y + threadIdx.y;
  
  //OBTÉM o ladrilho da COLUNA k na memória global e atribui para a memória compartilhada
  ladrilho[threadIdx.y][threadIdx.x] = F[tidY * n + tidX];

  //BARREIRA de sincronização - Garante que a memória compartilhada seja preenchida antes do processamento	
  __syncthreads();
  
  //PROCESSA o ladrilho da COLUNA k
  for(int m=0; m < blockDim.x; m++) {

    if(ladrilhoPrincipal[m][threadIdx.x] == 1 && ladrilho[threadIdx.y][m] == 1){
      ladrilho[threadIdx.y][threadIdx.x] = 1;   
    }        
	//BARREIRA de sincronização - Garante que as dependências de dados sejam satisfeitas. 
    __syncthreads();
  }
  
  //ATUALIZA os valores do ladrilho da COLUNA k na memória global
  F[tidY * n + tidX] = ladrilho[threadIdx.y][threadIdx.x];   
}

__host__
void pre_processamento(int* gpuF, int k, int size, dim3 grid, dim3 bloco){
    warshallKernelShared_Principal<<< 1, bloco  >>>(gpuF, k, size);
    warshallKernelShared_LinhaColuna<<< dim3(grid.x,1), bloco  >>>(gpuF, k, size);
}

void warshallCPU(int* fechoMatriz, unsigned n)
{		
		for(int k = 0; k < n; k++){
			for(int i = 0; i < n; i++){
				for(int j = 0; j < n; j++){
						if(fechoMatriz[k * n + j] == 1 && fechoMatriz[i * n + k] == 1)	
							fechoMatriz[i * n + j] = 1;
				}			
			}					
		}
}

void processamentoCPU(int *A, unsigned n)
{
	int* F = (int*) malloc( sizeof(int) * n * n);
  
  memcpy(F, A, sizeof(int)*n*n);

  double tempoGasto;
	clock_t start = clock();

	warshallCPU(F, n);
	
	clock_t stop = clock();
	
	tempoGasto = (stop - start) / (float) CLOCKS_PER_SEC;
 
	printf("Tempo de execução da CPU: %f s\n", tempoGasto ); 
  imprimeSoma(F, n);

	free(F);
}


void mainWarshall()
{

	int byteNumber = QTD_ELEMENTOS * QTD_ELEMENTOS * sizeof(int);

	int *A = (int*) malloc(byteNumber);
	
	inicializaMatriz(A, QTD_ELEMENTOS);
	processamentoCPU(A, QTD_ELEMENTOS);

	free(A);	
}

int main(void)
{
	mainWarshall();
	return 0;
}

