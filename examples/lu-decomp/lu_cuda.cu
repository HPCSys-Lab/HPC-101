#include "hip/hip_runtime.h"
%%cu

#define N 2048
#define NUM_THREADS 128
#include <stdio.h>
#include <time.h>
#include <assert.h>

typedef float* ptrmat; 
typedef float typemat; 


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

//--------------------------------------------------------------------------
void initMatrix(ptrmat matrix, unsigned size)
{
  time_t t;
	srand((unsigned int) time(&t));
	for (int i=0; i<size; i++) {
     for (int j=0; j<size; j++) { 
		   //matrix[i * size + j] = (typemat)( rand() & 0xFF )/10.0f;
			 matrix[i * size + j] = rand() % 50;			 
     }  
	}
}

void initMatrixL(ptrmat matrixL, unsigned size)
{
  
	for (int i=0; i<size; i++) {
     for (int j=0; j<size; j++) { 
       if(i == j )
			    matrixL[i * size + j] = 1;			 
     }  
	}
}


void multiMatrizesCPU(ptrmat A, ptrmat B, ptrmat C, unsigned n)
{
		typemat soma;
		for(int lin = 0; lin < n; lin ++){
				for(int col = 0; col < n; col ++){
						soma = 0;
						for(int k = 0; k < n; k++){
								soma += A[lin * n + k] * B[k * n + col];
						}
						C[lin * n + col] = soma;
				}					
		}
}


void imprimeSomaElementos(ptrmat matrix, unsigned n)
{
    double soma = 0;
    for (int i=0; i < n; i++) {
        for (int j=0; j < n; j++){
					soma += matrix[i * n + j];		
				}	    
	  } 
    printf("A soma dos elementos da matriz é %f\n",soma);
}

void checkDecompLU(ptrmat matrixL, ptrmat matrixU){

  int elementNumber  = N * N * sizeof(typemat);
  ptrmat C  = (ptrmat) malloc(elementNumber);

  multiMatrizesCPU(matrixL, matrixU, C, N);
  imprimeSomaElementos(C, N);

  free(C);
}

//--------------------------------------------------------

__global__ 
void parallel_algorithm_LU(ptrmat matrixL, ptrmat matrixU, int pivotStep, int n){

  int pivotIndex = pivotStep - 1;

  int i = blockIdx.y * blockDim.y + pivotStep  + threadIdx.y;
  int j = blockIdx.x * blockDim.x + pivotIndex + threadIdx.x;
  
  __shared__ typemat pivotValue;
  __shared__ double factor;

  if(threadIdx.x == 0 && threadIdx.y == 0){
      pivotValue = matrixU[pivotIndex * n + pivotIndex];
      if( pivotValue != 0 )
        factor = ((double) matrixU[i * n + pivotIndex]) / pivotValue;
      else
        printf("Não tem decomposição LU!!!\n");  
  }

  __syncthreads();

  if( (i >= n) ||  (j >= n) || pivotValue == 0 ) return;

  int indexGlobal = i * n + j;
    
  matrixU[ indexGlobal ] = matrixU[ indexGlobal ] - factor * matrixU[ pivotIndex * n + j];

  if(j == pivotIndex && i > pivotIndex)
  {
      matrixL[ indexGlobal ] = factor; 
  }
}

void decompLU_GPU(ptrmat matrix, ptrmat matrixL, ptrmat matrixU){

  int byteNumber = N * N * sizeof(typemat);

  ptrmat gpu_matrixU;
  ptrmat gpu_matrixL;

  //Define o tamanho das dimensões do bloco
  int blockSize_X = NUM_THREADS;
  int blockSize_Y = 1;
 
  //Define o tamanho das dimensões da grid
  int gridSize_X = ceil( ((float) N )/ blockSize_X);
  int gridSize_Y = ceil( ((float)  (N - 1) ) / blockSize_Y); 
  
  dim3 block = dim3(blockSize_X, blockSize_Y); 
  dim3 grid  = dim3(gridSize_X, gridSize_Y); 

  //Device memory allocation
  checkCuda( hipMalloc( (void**) &gpu_matrixU, byteNumber) );
  checkCuda( hipMalloc( (void**) &gpu_matrixL, byteNumber) );

  //Copy -- Host memory ---> Device memory
  checkCuda( hipMemcpy(gpu_matrixL, matrixL, byteNumber, hipMemcpyHostToDevice) );	
	checkCuda( hipMemcpy(gpu_matrixU, matrix, byteNumber, hipMemcpyHostToDevice) );	
 	
  hipEvent_t start, stop;
	float cpu_time = 0.0f;

	checkCuda( hipEventCreate(&start) );
	checkCuda( hipEventCreate(&stop) );

	checkCuda( hipEventRecord(start, 0) );
  
  //Launching kernels - DecompLU Execution
  for(int stepPivot = 1; stepPivot < N; stepPivot++){
          
    parallel_algorithm_LU<<<grid, block>>>(gpu_matrixL, gpu_matrixU, stepPivot, N);  
	
    grid.x = ceil( ((float) (N - stepPivot)) / blockSize_X);
    grid.y = grid.y - 1;

  // Obtém os erros de lançamento de kernel
	  checkCuda( hipGetLastError() );  
    
  }
  
	checkCuda( hipEventRecord(stop, 0) );
	checkCuda( hipEventSynchronize(stop) );
	checkCuda( hipEventElapsedTime(&cpu_time, start, stop) );

  //Copy -- Device memory ---> Host memory
  checkCuda( hipMemcpy(matrixL, gpu_matrixL, byteNumber, hipMemcpyDeviceToHost) );	
	checkCuda( hipMemcpy(matrixU, gpu_matrixU, byteNumber, hipMemcpyDeviceToHost) );
    
  //Free memory - Device
  checkCuda( hipFree(gpu_matrixU) );
  checkCuda( hipFree(gpu_matrixL) );

  printf("Tempo de Execução da GPU: %.4f ms \n", cpu_time);
}


int main(){
    
  ptrmat matrix;
  ptrmat matrixL;
  ptrmat matrixU;

  int elementNumber = N * N * sizeof(typemat);
  
  //Device memory allocation
  matrix  = (ptrmat) malloc(elementNumber);
  matrixL = (ptrmat) malloc(elementNumber);
  matrixU = (ptrmat) malloc(elementNumber);

  memset(matrixL, 0, elementNumber);
  initMatrixL(matrixL, N);
  
  initMatrix(matrix, N);
  imprimeSomaElementos(matrix, N);
 
  decompLU_GPU(matrix, matrixL, matrixU);

  checkDecompLU(matrixL, matrixU);

  //Free memory - Host 
  free(matrix);
  free(matrixU);
  free(matrixL);
  
  return 0;
}
